
#include "hip/hip_runtime.h"


#include <stdio.h>

int A[5] = { 1, 2, 3, 4, 5 };

__device__ int d_A[5];

__global__ void multiply()
{
	int i = threadIdx.x;
	d_A[i] = d_A[i] * 2;
}

int main()
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_A), A, 5 * sizeof(int));
	multiply <<< 1, 5 >>> ();
	hipMemcpyFromSymbol(A, HIP_SYMBOL(d_A), 5 * sizeof(int));

	for (int i = 0; i < 5; i++)
	{
		printf("%d ", A[i]);
	}
}